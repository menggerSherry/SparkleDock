#include "cal_anm.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
// #include "error.cuh"
#include <iostream>
#include <unistd.h>


#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUSOLVER(call) { \
    hipsolverStatus_t err = call; \
    if (err != HIPSOLVER_STATUS_SUCCESS) { \
        std::cerr << "cuSolver error in " << __FILE__ << " at line " << __LINE__ << ": " << err << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}



void eigh_gpu(double * eigenvalues, double * eigenvectors, double * matrix, int * eigvals, int n, int rank, int size){

    int gpu_device = rank % 4;
    CHECK_CUDA(hipSetDevice(gpu_device));

    hipsolverHandle_t cusolver_handle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolver_handle));

    int lda = n;
    double *d_eigenvalues = nullptr, *d_matrix = nullptr, *d_work = nullptr;
    int *d_info = nullptr, info = 0, lwork = 0;

    double *eigenvectors_buffer = (double *)malloc(n * n * sizeof(double));

    CHECK_CUDA(hipMalloc((void**)&d_matrix, n * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_eigenvalues, n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice));

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // Compute eigenvalues & eigenvectors
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolver_handle, jobz, uplo, n, d_matrix, lda, d_eigenvalues, &lwork));
    lwork = static_cast<int>(lwork * 1.5);  

    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    // Compute eigenvalues
    CHECK_CUSOLVER(hipsolverDnDsyevd(cusolver_handle, jobz, uplo, n, d_matrix, lda, d_eigenvalues, d_work, lwork, d_info));

    // 确保计算完成
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info != 0) {
        std::cerr << "cuSolver failed on rank " << rank << ", info = " << info << std::endl;
        exit(EXIT_FAILURE);
    }

    int start = eigvals[0], end = eigvals[1], length = end - start + 1;
    CHECK_CUDA(hipMemcpy(eigenvectors_buffer, d_matrix, n * n * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(eigenvalues, d_eigenvalues + start, length * sizeof(double), hipMemcpyDeviceToHost));

    // 释放资源
    CHECK_CUDA(hipFree(d_matrix));
    CHECK_CUDA(hipFree(d_eigenvalues));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolver_handle));
    CHECK_CUDA(hipDeviceReset());

    // 重新整理 eigenvectors
    for (int i = 0; i < length; i++) {
        for (int j = 0; j < n; j++) {
            eigenvectors[j * length + i] = eigenvectors_buffer[i * n + j];
        }
    }
    free(eigenvectors_buffer);
    

}





